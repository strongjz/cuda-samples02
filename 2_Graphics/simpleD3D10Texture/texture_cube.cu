/*
 * Copyright 1993-2013 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define PI 3.1415926536f

/*
 * Paint a 2D surface with a moving bulls-eye pattern.  The "face" parameter selects
 * between 6 different colors to use.  We will use a different color on each face of a
 * cube map.
 */
__global__ void cuda_kernel_texture_cube(char *surface, int width, int height, size_t pitch, int face, float t)
{
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned char *pixel;

    // in the case where, due to quantization into grids, we have
    // more threads than pixels, skip the threads which don't
    // correspond to valid pixels
    if (x >= width || y >= height) return;

    // get a pointer to this pixel
    pixel = (unsigned char *)(surface + y*pitch) + 4*x;

    // populate it
    float theta_x = (2.0f*x)/width  - 1.0f;
    float theta_y = (2.0f*y)/height - 1.0f;
    float theta = 2.0f*PI*sqrt(theta_x*theta_x + theta_y*theta_y);
    unsigned char value = 255*(0.6f + 0.4f*cos(theta + t));

    pixel[3] = 255; // alpha

    if (face%2)
    {
        pixel[0] =    // blue
            pixel[1] =    // green
                pixel[2] = 0.5; // red
        pixel[face/2] = value;
    }
    else
    {
        pixel[0] =        // blue
            pixel[1] =        // green
                pixel[2] = value; // red
        pixel[face/2] = 0.5;
    }
}

extern "C"
void cuda_texture_cube(void *surface, int width, int height, size_t pitch, int face, float t)
{
    hipError_t error = hipSuccess;

    dim3 Db = dim3(16, 16);   // block dimensions are fixed to be 256 threads
    dim3 Dg = dim3((width+Db.x-1)/Db.x, (height+Db.y-1)/Db.y);

    cuda_kernel_texture_cube<<<Dg,Db>>>((char *)surface, width, height, pitch, face, t);

    error = hipGetLastError();

    if (error != hipSuccess)
    {
        printf("cuda_kernel_texture_cube() failed to launch error = %d\n", error);
    }
}

